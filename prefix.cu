#include "hip/hip_runtime.h"
%%writefile hello.cu
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

// Kernel for parallel prefix sum
__global__ void prefixSum(int *input, int *output, int n) {
    extern __shared__ int temp[];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;

    // Load input into shared memory
    temp[tid] = (idx < n) ? input[idx] : 0;
    __syncthreads();

    // Reduction phase
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        int index = 2 * stride * (tid + 1) - 1;
        if (index < blockDim.x) {
            temp[index] += temp[index - stride];
        }
        __syncthreads();
    }

    // Downsweep phase
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        int index = 2 * stride * (tid + 1) - 1;
        if (index + stride < blockDim.x) {
            temp[index + stride] += temp[index];
        }
        __syncthreads();
    }

    // Store result to output
    if (idx < n) {
        output[idx] = temp[tid];
    }
}

// Wrapper function for inclusive prefix sum calculation
void inclusivePrefixSum(int *d_input, int *d_output, int n) {
    int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start time
    hipEventRecord(start);

    // Perform parallel prefix sum
    prefixSum<<<numBlocks, BLOCK_SIZE, BLOCK_SIZE * sizeof(int)>>>(d_input, d_output, n);
    hipDeviceSynchronize();

    // Record stop time
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

   std::cout << "Execution Time: " << milliseconds * 1000 << " microseconds" << std::endl;
}

int main() {
    int n = 5; // Size of the array (assuming 32n elements)
    int arraySize = 32 * n;

    // Host array
    int *h_input = new int[arraySize];

    // Seed for random number generation
    srand(time(nullptr));

    // Generate random numbers for the array
    for (int i = 0; i < arraySize; ++i) {
        h_input[i] = rand() % 100; //
    }

    // Device arrays
    int *d_input, *d_output;
    hipMalloc(&d_input, arraySize * sizeof(int));
    hipMalloc(&d_output, arraySize * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_input, h_input, arraySize * sizeof(int), hipMemcpyHostToDevice);

    // Perform inclusive prefix sum
    inclusivePrefixSum(d_input, d_output, arraySize);

    // Copy result back to host
    int *h_output = new int[arraySize];
    hipMemcpy(h_output, d_output, arraySize * sizeof(int), hipMemcpyDeviceToHost);

    // Print results
    std::cout << "Input array: ";
    for (int i = 0; i < arraySize; ++i) {
        std::cout << h_input[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "Inclusive Prefix Sum: ";
    for (int i = 0; i < arraySize; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    // Clean up
    delete[] h_input;
    delete[] h_output;
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}

